#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

#define BLOCK_DIM 16
__global__ void matching(int *pic, int* object,double* matchMatrix, double match, int picDim, int objDim, int picId, int objectId)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
    	int row = blockIdx.y * blockDim.y + threadIdx.y;
   	int k = col + row * picDim;//the point in the matrix
   	double sum = 0;								
   	if (k + objDim < picDim*picDim && col+objDim <= picDim && row+objDim <= picDim) //make sure the point is in the matrix
   	{	
   		for(int i=0;i<objDim;i++)//pass through all the points in the matching area										
   		{
   			for(int j=0;j<objDim;j++)
   			{
   				double tmpValue = ((double)pic[k+(i*picDim) + j] - (double)object[i*objDim+j])/(double)pic[k+(i*picDim) + j];
   				
   				if(tmpValue<0)
   					tmpValue = tmpValue * -1;
   				sum += tmpValue;
   			}
   		}
   		int mtachMatDim = picDim - objDim + 1;
   		
   		matchMatrix[(k/picDim)*mtachMatDim +  k%picDim] = sum;//put the result in the results matrix
    	}
}

int computeOnGPU(int *pic,int* object,double* matchMatrix,int picDim,int objectDim,int picId,int objectId, double match)
{
	hipError_t err = hipSuccess;
	int sizePic = picDim*picDim;
	int sizeObject = objectDim*objectDim;
	int matchMatrixDim = picDim - objectDim + 1;
	int *d_A, *d_B;
	double *d_MatchMatrix;
	err = hipMalloc((void **)&d_A, sizePic*sizeof(int));
    	if (err != hipSuccess) {
        	fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
       		 exit(EXIT_FAILURE);
    	}
    	err = hipMalloc((void **)&d_B, sizeObject*sizeof(int));
    	if (err != hipSuccess) {
        	fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
       		 exit(EXIT_FAILURE);
    	}
    	err = hipMalloc((void **)&d_MatchMatrix, matchMatrixDim*matchMatrixDim*sizeof(double));
    	if (err != hipSuccess) {
        	fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
       		 exit(EXIT_FAILURE);
    	}
    	err = hipMemcpy(d_A, pic, sizePic*sizeof(int), hipMemcpyHostToDevice);
   	 if (err != hipSuccess) {
       	 	fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
    	err = hipMemcpy(d_B, object, sizeObject*sizeof(int), hipMemcpyHostToDevice);
   	 if (err != hipSuccess) {
       	 	fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
    	err = hipMemcpy(d_MatchMatrix, matchMatrix, matchMatrixDim*matchMatrixDim*sizeof(double), hipMemcpyHostToDevice);
   	 if (err != hipSuccess) {
       	 	fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
	dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    	dim3 dimGrid((picDim+dimBlock.x-1)/dimBlock.x, (picDim+dimBlock.y-1)/dimBlock.y);

    	
    	matching<<<dimGrid,dimBlock>>>(d_A, d_B,d_MatchMatrix, match, picDim, objectDim, picId, objectId);
    	
    	err = hipGetLastError();
    	if (err != hipSuccess) {
        	fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
    	err = hipMemcpy(matchMatrix, d_MatchMatrix, matchMatrixDim*matchMatrixDim*sizeof(double), hipMemcpyDeviceToHost);
    	if (err != hipSuccess) {
      	  fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
     	   exit(EXIT_FAILURE);
   	 }

    	// Free allocated memory on GPU
   	 if (hipFree(d_MatchMatrix) != hipSuccess) {
   	     fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
   	     exit(EXIT_FAILURE);
   	 }
   	 if (hipFree(d_A) != hipSuccess) {
   	     fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
   	     exit(EXIT_FAILURE);
   	 }
   	 if (hipFree(d_B) != hipSuccess) {
   	     fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
   	     exit(EXIT_FAILURE);
   	 }
    	return 0;
}
